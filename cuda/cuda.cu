#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>  // for generationg random values in GPU
#include "hpc.h"

// nvcc cuda.cu -o cuda
// ./cuda <n_input_neurons> <R> <n_layers>
// ./cuda 1000000 3 100

const float bias = 0.1; // Constant bias 

// Sigmoid function, simple version 
__device__ float sigmoid(float x) {
    return 1.0 / (1.0 + expf(-x));
}

#define __cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Kernel that calculates a sone output values of next output layer 
__global__ void compute_layerGPU(  float *activations,   
                                float *weights,  
                                int next_layer_size,  
                                int R,      
                                int activations_offset,
                                int weights_offset,
                                int output_offset   // index of the first output neuron  
                             ) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;    // index of the output neuron 
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < next_layer_size; i += stride)
    {
        // printf("Thread idx = %d passed\n", idx);
        float sum = 0.0;
        for (int r = 0; r < R; r++) {
            sum += activations[activations_offset + i + r] * weights[weights_offset + (i * R) + r];
        }
        // printf("Thread idx = %d total sum: %.6f \n", idx, sum);
        activations[output_offset + i] = sigmoid(sum + bias);
    }
}

// initalizate an array with random float values (0, 1) range
__global__ void initializeRandomArray(float *array, int size, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);  // Initialize the RNG with a seed
        array[idx] = hiprand_uniform(&state);  // Generate a random float in [0, 1)
    }
}

int main(int argc, char *argv[]) {

    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    printf("Number of SMs: %d\n", numberOfSMs);

    float tstart, tstop;
    long tot_number_of_bytes_allocated = 0;

    if (argc != 4) {
        printf("Usage: %s <N> <R> <K>\n", argv[0]);
        return -1;
    }

    // Read input params 
    int N = atoi(argv[1]);
    int R = atoi(argv[2]);
    int K = atoi(argv[3]);
    printf("N=%d, R=%d, K=%d\n", N, R, K);

    // Compute total number of weights 
    int total_weights = 0;
    int layer_size;
    int total_neurons = N;   // input layer has N neurons 
    for (int t = 1; t < K ; t++) {   // we have weights for K-1 layers (we don't have weights for the input )
        layer_size = N - t * (R - 1);   // numbers of neurons for the current layer 
        total_neurons += layer_size; // update the number of total neurons
        total_weights += layer_size * R;    // we are R unique weights for each neuron
    }
    printf("Output layer size: %d\n", layer_size);
    printf("Total number of weigths: %d\n", total_weights);

    // Data allocation on CPU 
    // we want to allocate two large sequential arrays, one for neurons activation 
    // and one for weights 
    int size = sizeof(float);   //both weights and activation are float's

    // printf("CPU allocation...\n");
    tstart = hpc_gettime();
    float *activationsCPU = (float *)malloc(total_neurons * size);
    float *weightsCPU = (float *)malloc(total_weights * size);
    tstop = hpc_gettime();
    
    tot_number_of_bytes_allocated += (K * sizeof(float *)) + (total_weights * sizeof(float));

    // GPU allocation and initialization 
    float *activationsGPU, *weightsGPU;
    __cudaCheckError(hipMalloc(&activationsGPU, total_neurons * size));
    __cudaCheckError(hipMalloc(&weightsGPU, total_weights * size));
    
    // Launch kernel to initialize weights with random values
    int seed = 99;
    int threadsPerBlock = 1024;
    int blocksPerGrid = (total_weights + threadsPerBlock - 1) / threadsPerBlock;

    tstart = hpc_gettime();
    initializeRandomArray<<<blocksPerGrid, threadsPerBlock>>>(weightsGPU, total_weights, seed);
    __cudaCheckError( hipPeekAtLastError() );
    __cudaCheckError( hipDeviceSynchronize() );
    initializeRandomArray<<<blocksPerGrid, threadsPerBlock>>>(activationsGPU, N, seed);
    __cudaCheckError( hipPeekAtLastError() );
    __cudaCheckError( hipDeviceSynchronize() );

    // Copy data back from GPU to CPU
    __cudaCheckError(hipMemcpy(activationsCPU, activationsGPU, total_neurons * sizeof(float), hipMemcpyDeviceToHost));
    __cudaCheckError(hipMemcpy(weightsCPU, weightsGPU, total_weights * sizeof(float), hipMemcpyDeviceToHost));
    tstop = hpc_gettime();
    printf("Preapration time: %.10f\n", tstop - tstart);
    // Verify by printing fisrt and last 10 values of weights
    // for (int i = total_weights - 5; i < total_weights; i++) {
    //     printf("weightsCPU[%d] = %f \n", i, weightsCPU[i]);
    // } 
    // printf("\n");
    // for (int i = total_weights - 5; i < total_weights; i++) {
    //     printf("weightsCPU[%d] = %f \n", i, weightsCPU[i]);
    // }
    // printf("\n\n");
        // Verify by printing fisrt and last 10 values of weights
    // for (int i = 0; i < total_neurons; i++) {
    //     printf("activationsCPU[%d] = %f \n", i, activationsCPU[i]);
    // } 
    // printf("\n");

    printf("\n\n");


    int activations_offset = 0;
    int weights_offset = 0;
    tstart = hpc_gettime();
    for (int t = 1; t < K; t++) {   // we iterate from layer 1 to layer K-1
        int input_layer_size = N - (t-1) * (R - 1);   // input layer size
        int output_layer_size = N - t * (R - 1);  // output layer size

        int output_idx = activations_offset + input_layer_size;    

        // Numero di thread per blocco
        int numBlocks = (output_layer_size + threadsPerBlock - 1) / threadsPerBlock;
        // printf("Lunching:\n %d blocks of %d threads each. \n Toatal: %d\n", numBlocks, threadsPerBlock, numBlocks * threadsPerBlock);
        // Chiamata al kernel CUDA
        compute_layerGPU<<<numBlocks, threadsPerBlock>>>(activationsGPU, weightsGPU, output_layer_size, R, activations_offset, weights_offset, output_idx);
        __cudaCheckError( hipPeekAtLastError() );
        __cudaCheckError( hipDeviceSynchronize() );
    
        // update the activation offset at the first neuron of the next input layer
        activations_offset += input_layer_size;
        weights_offset += output_layer_size * R;
    }
    tstop = hpc_gettime();
    printf("Compute time GPU: %.10f\n", tstop - tstart);

    // Copy data back from GPU to CPU
    __cudaCheckError(hipMemcpy(activationsCPU, activationsGPU, total_neurons * sizeof(float), hipMemcpyDeviceToHost));

    // Verify by printing last 10 values of activations
    for (int i = total_neurons - 10; i < total_neurons; i++) {
        printf("activationsCPU[%d] = %f\n ", i, activationsCPU[i]);
    }

    // Deallocazione memoria
    free(activationsCPU);
    free(weightsCPU);
    
    // Deallocazione memoria sulla GPU
    hipFree(activationsGPU);
    hipFree(weightsGPU);
}